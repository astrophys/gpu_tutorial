/*
Author : Ali Snedden
Date   : 8/21/18
Purpose:
    This is a program that multiplies two matrices.
Debug  : 
Notes  : 
    1. How to Run:
        module load cuda/8.0 
        nvcc matrix_multiply.cu    
    2. http://developer.download.nvidia.com/compute/cuda/3_1/toolkit/docs/NVIDIA_CUDA_C_ProgrammingGuide_3.1.pdf
    3. Unified Memory : https://devblogs.nvidia.com/unified-memory-cuda-beginners/
    4. Cannot kill errant threads and cleanly end computation in CUDA
       --> See : https://stackoverflow.com/questions/52116815/is-there-a-way-terminate-host-and-device-program-execution-if-a-cuda-thread-enco
    5. 'Proper' error handling : https://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api
    6. Recall that we can't print to stderr from gpu thread
    7. To debug cuda-gdb ./a.out
    8. Biggest error is in forgetting to allocate memory between the device and host,
       e.g. cudaMallocManaged()
    9. IO is very expensive.  Appears to get flushed on cudaDeviceSynchronize()
    10. Using matrix_multiply<<<1,1>>> == 431s, while cpu version 14s. Clearly there
        is substantial overhead when using running single gpu thread.
    11. If using more than maxThreadsPerBlock, it fails to compute and doesn't emit an 
        error.
        --> After each kernel call, do gpuErrchk( cudaPeekAtLastError() );

Good Weblinks:
    1. Unified Memory : https://devblogs.nvidia.com/unified-memory-cuda-beginners/

Future :
    1. Try managing memory directly on Host and Device.

*/

#include <hip/hip_runtime.h>
#include <iostream>
#include <string.h>
#include <stdio.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime_api.h>

// This is C++ code - from stackoverflow : https://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api
/********************************************************
    ARGS:
        cudaError_t code
        const char* file : 
        int line :
    DESCRIPTION:
        Uses macro and inline function b/c it is important to preserve the
        file and line number in the error printing.
    RETURN:
    DEBUG:
    NOTES: 
    FUTURE:
*******************************************************/
#define gpuErrChk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s : %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}



/**********************************
ARGS:
RETURN:
DESCRIPTION:
    Map 2D indices to 1D index
DEBUG:
    1. read_numpy_matrix() uses this function extensively.
       Directly compared output from read_numpy_matrix() with input
       and was IDENTICAL. This could not work if map_idx() didn't 
       function correctly.
FUTURE:
    1. Add error checking if not too expensive
***********************************/
int map_idx(int i, int j, int Ny){
    return (Ny * i + j);
}
__device__ int d_map_idx(int i, int j, int Ny){
    return (Ny * i + j);
}

/********************************************************
    ARGS:
    DESCRIPTION:
    RETURN:
    DEBUG:
    NOTES: 
    FUTURE:
*******************************************************/
void exit_with_error(char * message){
    fprintf(stderr, "%s", message);
    fflush(stderr);
    exit(1);
}
//__device__ void d_exit_with_error(char * message){
//    fprintf(stderr, "%s", message);
//    fflush(stderr);
//    exit(1);
//}



/**********************************
ARGS:
    path = path to file to read
    dim  = dimension of returned matrix, expected to be len = 2
RETURN:
DESCRIPTION:
    Map 2D indices to 1D index
DEBUG:
    1. Printed out read in matrix. used 'diff' to compare 
       with original. Was IDENTICAL
       --> This function WORKS!
FUTURE:
    1. Add error checking if not too expensive
***********************************/
float * read_numpy_matrix(char* path, int * dim){
    char * line= NULL;
    char * entireFile = NULL;
    char * pch = NULL;  // Used for parsing strings w strtok
    char errStr[500];
    int fileSize = -1;
    int nline = 0;
    int maxchar = 0;    // Maximum number of characters in a lines
    int nchar = 0;      // Number of characters in line
    int ncols = -1;     // Ncolumns in each row.. should be the same for each row
    int ncolsThisRow = 0;   
    int i = 0;
    int j = 0;
    int n = 0;          // Index to loop thru _all_ file chars
    float * matrix = NULL;
    FILE * f = fopen(path, "r");

    printf("\treading : %s\n", path);
    fflush(stdout);

    //Error check
    if(f == NULL){
        sprintf(errStr, "ERROR!!! %s cannot be opened", path);
        exit_with_error(errStr);
    }
    //Get file size
    fseek(f, 0, SEEK_END);
    fileSize = ftell(f);    // Total num chars in file
    rewind(f);

    //Read entire file
    entireFile = (char* )malloc(sizeof(char) * fileSize);
    fread(entireFile, sizeof(char), fileSize, f);
    rewind(f);

    //Find number of lines and maxchar per line...
    for(n=0; n<fileSize; n++){
        if(entireFile[n] == ' '){
            ncolsThisRow++;
        }
        
        if(entireFile[n] == '\n'){
            maxchar = nchar > maxchar ? nchar : maxchar;

            //Must set at first
            if(nline == 0){
                ncols = ncolsThisRow;
            //Oops, rows aren't the same size.
            }else if(ncols != ncolsThisRow){
                sprintf(errStr, "ERROR!!! nchar %i != ncolsThisRow %i\n", nchar, ncolsThisRow);
                exit_with_error(errStr);
            }
            ncolsThisRow=0;
            nchar = 0;
            nline++;
        }
        nchar++;
    }
    maxchar = maxchar + 1; //+1 for null terminator?
    printf("dim = [nline, ncols] =  [%i, %i],  maxchar = %i \n", nline, ncols, maxchar);
    fflush(stdout);
    
    // Done with busy work - now allocate memory, read in array
    hipMallocManaged(&matrix, nline * maxchar * sizeof(float));
    line   = (char *)malloc(sizeof(char) * maxchar);
    i = 0;
    while(feof(f) == 0){
        if(fgets(line, maxchar, f)){
            //printf("\tEnd of File Reached\n\n");
            //sprintf(errStr, "ERROR!!! in reading 'line'\n");
            //exit_with_error(errStr);
        }
        // Parse line in file
        pch = strtok(line," ");
        j = 0;
        while(pch != NULL){
            matrix[map_idx(i,j,ncols)] = (float)atof(pch);
            pch = strtok(NULL, " ");
            j++;
        }
        i++;
    }

    /* Debug 
    for(i=0; i<nline; i++){
        for(j=0; j<ncols; j++){
            printf("%.1f ", matrix[map_idx(i,j,ncols)]);
        }
        printf("\n");
    }*/
    
    free(line);
    free(entireFile);
    fclose(f);
    dim[0] = nline;
    dim[1] = ncols;
    return matrix;
}
 
 

/**********************************
ARGS:
    array1D : 'flattened' 2D array as 1D
    N       : length of array
RETURN:
    N/A
DESCRIPTION:
    Prints 1D array and 3D coords
DEBUG:
    1. spot checked, it works
FUTURE:
***********************************/
void write_1D_array(float * array1D, int Nx, int Ny, FILE * f){
    int i = 0;
    int j = 0;
    int idx = 0;
    for(i=0; i<Nx; i++){
        for(j=0; j<Ny; j++){
            idx = map_idx(i,j,Ny);
            fprintf(f, "%*.1f ", 5, array1D[idx]);
        }
        fprintf(f, "\n");
    }
}


/**********************************
ARGS:
    array1D : 'flattened' 2D array as 1D
    N       : length of array
RETURN:
    N/A
DESCRIPTION:
    Prints 1D array and 3D coords
DEBUG:
    1. spot checked, it works
FUTURE:
***********************************/
void print_1D_array(float * array1D, int Nx, int Ny){
    int i = 0;
    int j = 0;
    int idx = 0;
    for(i=0; i<Nx; i++){
        for(j=0; j<Ny; j++){
            idx = map_idx(i,j,Ny);
            printf("%*.1f ", 5, array1D[idx]);
        }
        printf("\n");
    }
}

/********************************************************
    ARGS:
    DESCRIPTION:
    RETURN:
    DEBUG:
    NOTES: 
        1. Use 'flattened' 2D array
    FUTURE:
*******************************************************/
void initialize_matrix(float *A, int * dim, float value){
    for(int i=0; i<dim[0]; i++){
        for(int j=0; j<dim[1]; j++){
            //A[i*dim[0]+j] = value;
            A[map_idx(i,j,dim[1])] = value;
        }       
    }

}


/********************************************************
    ARGS:
        A : 'flattened' 2d matrix
        B : 'flattened' 2d matrix
        dimA : gives x & y dims
        dimB : gives x & y dims
        dimAB: pointer modified to return size of new matrix

    DESCRIPTION:
        Multiply A*B : Check dims. Expect only 2 dimensions
        for dimA and dimB.
    RETURN:
    DEBUG:
        1. created code, matrix_generator.py, that multiplies two matrices and
           saves the input and output to a file. I read in data/A.txt, data/B.txt
           and used this function to multiply the matrices. Printed the output and 
           compared to data/AB.txt. It was IDENTICAL. 
           --> This function works!
    NOTES: 
    FUTURE:
*******************************************************/
float * cpu_matrix_multiply(float * A, float * B, int * dimA, int * dimB, int * dimAB)
{
    int j = 0;          // Iterate over elements, do dot product
    int ai = 0;         // Index iterating over rows in A
    int bj = 0;         // Index iterating over columns in B
    float sum = 0;
    char errStr[500];
    float * result = (float *)malloc(sizeof(float) * dimA[0] * dimB[1]);

    // Error Check
    if(dimA[1] != dimB[0]){
        sprintf(errStr, "ERROR!! dimension mismatch, %i != %i", dimA[1], dimB[0]);
        exit_with_error(errStr);
    }

    for(ai=0; ai<dimA[0]; ai++){
        for(bj=0; bj<dimB[1]; bj++){
            sum = 0;
            for(j=0; j<dimA[1]; j++){
                //printf("%.0f * %0.f\n", A[map_idx(ai, j, dimA[1])],
                //        B[map_idx(j, bj, dimB[1])]);

                sum += A[map_idx(ai, j, dimA[1])] * B[map_idx(j, bj, dimB[1])];
                result[map_idx(ai,bj,dimB[1])] = sum;
            }
            //printf("\n");
        }
    }
    dimAB[0] = dimA[0];
    dimAB[1] = dimB[1];
    return result;
}


/********************************************************
    ARGS:
        A : 'flattened' 2d matrix
        B : 'flattened' 2d matrix
        dimA : gives x & y dims
        dimB : gives x & y dims
        dimAB: pointer modified to return size of new matrix

    DESCRIPTION:
        Multiply A*B : Check dims. Expect only 2 dimensions
        for dimA and dimB.
    RETURN:
    DEBUG:
    NOTES: 
        blockDim.x  : number of threads in each block
        blockIdx.x  : index of current block
        threadIdx.x : 
    FUTURE:
*******************************************************/
__global__ void matrix_multiply(float * A, float * B, int * dimA, int * dimB,
                                  float * AB, int * dimAB)
{
    int j = 0;          // Iterate over elements, do dot product
    int startIdx = blockIdx.x * blockDim.x + threadIdx.x; // Index of current thread in block
    int stride   = blockDim.x * gridDim.x;                // Number of threads in the block
    int ai = 0;         // Index iterating over rows in A
    int bj = 0;         // Index iterating over columns in B
    float sum = 0;
    //printf("%i %i : [%i %i] %i %i\n", startIdx, stride, threadIdx.x, blockIdx.x, blockDim.x, gridDim.x);

    // Error Check - not possible on device code
    // if(dimA[1] != dimB[0]){
    //     char errStr[] = "ERROR!! dimension mismatch\n";
    //     //sprintf(errStr, "ERROR!! dimension mismatch, %i != %i", dimA[1], dimB[0]);
    //     d_exit_with_error(errStr);
    // }
    
    // Grid-stride loop
    for(ai=startIdx; ai<dimA[0]; ai+=stride){
        //printf("[%i %i] : %i : dimA[0] = %i\n", threadIdx.x, blockIdx.x, ai, dimA[0]);
        for(bj=0; bj<dimB[1]; bj++){
            sum = 0;
            for(j=0; j<dimA[1]; j++){
                //printf("\t[%i, %i] x [%i, %i]\n", ai, j, j, bj);  // EXPENSIVE!! increases runtime 100x

                sum += A[d_map_idx(ai, j, dimA[1])] * B[d_map_idx(j, bj, dimB[1])];
                AB[d_map_idx(ai,bj,dimB[1])] = sum;
            }
            //printf("\n");
        }
    }
}





/********************************************************
    ARGS:
        int argc        : 
        char *argv[]    : 
    DESCRIPTION:
        Can run as 
            ./a.out 
            ./a.out ouputfile
    RETURN:
    DEBUG:
    NOTES: 
    FUTURE:
*******************************************************/
int main(int argc, char *argv[])
{
    // Declare variables
    char path[100];
    char errStr[200];
    int nDev = 0;      //Number of devices
    int * dimA = NULL; //{2,3};
    int * dimB = NULL; //{3,2};
    int * dimAB = NULL; //{0,0};    // Initialize to some value
    float *A = NULL;
    float *B = NULL;
    float *AB = NULL;
    float *answer = NULL;
    FILE * fout = NULL;
    // Print device statistics.
    hipGetDeviceCount(&nDev);
    for(int i=0; i<nDev; i++){
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printf("Device : %i, Card : %s\n",i,prop.name);
        printf("warpSize : %d\n", prop.warpSize);
        printf("multiProcessorCount : %d\n", prop.multiProcessorCount);
        printf("maxThreadsPerMultiProcessor : %i\n", prop.maxThreadsPerMultiProcessor);
        printf("maxThreadsPerBlock : %i\n", prop.maxThreadsPerBlock);
        printf("maxGridSize : %i\n", prop.maxGridSize);
    }


    // This uses CUDA's Unified Memory
    gpuErrChk(hipMallocManaged(&dimA, 2 * sizeof(float)));
    gpuErrChk(hipMallocManaged(&dimB, 2 * sizeof(float)));
    gpuErrChk(hipMallocManaged(&dimAB, 2 * sizeof(float)));
    // Set dimensions
    dimA[0] = 2;
    dimA[1] = 3;
    dimB[0] = 3;
    dimB[1] = 2;
    gpuErrChk(hipMallocManaged(&A, dimA[0] * dimA[1] * sizeof(float)));
    gpuErrChk(hipMallocManaged(&B, dimB[0] * dimB[1] * sizeof(float)));
    
    // Initialize
    initialize_matrix(A,dimA,2);
    initialize_matrix(B,dimB,4);
    // Set my own values
    A[map_idx(0,0,dimA[1])] = 1;
    A[map_idx(0,1,dimA[1])] = 2;
    A[map_idx(0,2,dimA[1])] = 3;
    A[map_idx(1,0,dimA[1])] = 4;
    A[map_idx(1,1,dimA[1])] = 5;
    A[map_idx(1,2,dimA[1])] = 6;

    B[map_idx(0,0,dimB[1])] = 7;
    B[map_idx(0,1,dimB[1])] = 10;
    B[map_idx(1,0,dimB[1])] = 8;
    B[map_idx(1,1,dimB[1])] = 11;
    B[map_idx(2,0,dimB[1])] = 9;
    B[map_idx(2,1,dimB[1])] = 12;
    
    AB = cpu_matrix_multiply(A, B, dimA, dimB, dimAB);

    // Print matrices
    printf("Multiplying Trivial Matrices\n");
    printf("A (%i x %i):\n", dimA[0], dimA[1]);
    print_1D_array(A, dimA[0], dimA[1]);
    printf("B (%i x %i):\n", dimB[0], dimB[1]);
    print_1D_array(B, dimB[0], dimB[1]);
    printf("AB (%i x %i):\n", dimAB[0], dimAB[1]);
    print_1D_array(AB, dimAB[0], dimAB[1]);
    

    // Read matrix files
    gpuErrChk(hipFree(A));
    gpuErrChk(hipFree(B));
    free(AB); 
    //sprintf(path, "data/A_small.txt");
    sprintf(path, "data/A.txt");
    A = read_numpy_matrix(path, dimA);
    //sprintf(path, "data/B_small.txt");
    sprintf(path, "data/B.txt");
    B = read_numpy_matrix(path, dimB);
    //sprintf(path, "data/AB_small.txt");
    sprintf(path, "data/AB.txt");
    answer = read_numpy_matrix(path, dimAB);
    //AB = cpu_matrix_multiply(A, B, dimA, dimB, dimAB);
    //print_1D_array(AB, dimAB[0], dimAB[1]);

    // Try CUDA version of matrix_multiply
    dimAB[0] = dimA[0];
    dimAB[1] = dimB[1];
    gpuErrChk(hipMallocManaged(&AB, dimAB[0] * dimAB[1] * sizeof(float)));
    //            <<<gridDim.x (# blocks), blockDim.x (# threads per block) >>>

    time_t start = time(NULL);
    matrix_multiply<<<1024,32>>> (A, B, dimA, dimB, AB, dimAB);  // Fails b/c maxThreadsPerBlock=1024
    gpuErrChk( hipPeekAtLastError());
    gpuErrChk(hipDeviceSynchronize());

    printf("Run time : %.3f s\n", difftime(time(NULL), start));
    if(argc == 1){
        fout = fopen("output/AB_result.txt", "w+");
    }else if(argc == 2){
        fout = fopen(argv[1], "w+");
    }else{
        sprintf(errStr, "ERROR!!! Incorrect number of arguments");
        exit_with_error(errStr);
    }
    write_1D_array(AB, dimAB[0], dimAB[1], fout);
    fclose(fout);


    return 0;
}

