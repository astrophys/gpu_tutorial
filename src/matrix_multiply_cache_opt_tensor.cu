/*
Author : Ali Snedden
Date   : 11/28/18
Purpose:
    This is a program that multiplies two matrices. This combines features from 
    matrix_multiply_omp_cache_optimized.cu (e.g. taking the transpose of B) and using
    CUDA. This code uses the v100 tensor cores.
Debug  : 
Notes  : 
    1. How to Run:
        module load cuda/9.0 
        nvcc --gpu-architecture=compute_70 matrix_multiply.cu    
    2. Recall that we can't print to stderr from gpu thread
    3. To debug cuda-gdb ./a.out
    4. Biggest error is in forgetting to allocate memory between the device and host,
       e.g. cudaMallocManaged()
    5. IO is very expensive.  Appears to get flushed on cudaDeviceSynchronize()
    6. Using matrix_multiply<<<1,1>>> == 431s, while cpu version 14s. Clearly there
        is substantial overhead when using running single gpu thread.
    7. If using more than maxThreadsPerBlock, it fails to compute and doesn't emit an 
        error.
        --> After each kernel call, do gpuErrchk( cudaPeekAtLastError() );

Good Weblinks:
    1. Unified Memory : https://devblogs.nvidia.com/unified-memory-cuda-beginners/
    2. Tensor Core example : https://devblogs.nvidia.com/programming-tensor-cores-cuda-9/
    3. http://developer.download.nvidia.com/compute/cuda/3_1/toolkit/docs/NVIDIA_CUDA_C_ProgrammingGuide_3.1.pdf
    4. 'Proper' error handling : https://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api
    5. Cannot kill errant threads and cleanly end computation in CUDA
       --> See : https://stackoverflow.com/questions/52116815/is-there-a-way-terminate-host-and-device-program-execution-if-a-cuda-thread-enco
    6. Warp Matrix Functions : https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#wmma

Future :
    1. Try managing memory directly on Host and Device.

*/

#include <hip/hip_runtime.h>
#include <iostream>
#include <string.h>
#include <stdio.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime_api.h>

// This is for tensor cores...
#include <mma.h>
using namespace nvcuda; 
using namespace std; 
const int WMMA_M = 16;
const int WMMA_N = 16;
const int WMMA_K = 16;



// This is C++ code - from stackoverflow : https://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api
/********************************************************
    ARGS:
        cudaError_t code
        const char* file : 
        int line :
    DESCRIPTION:
        Uses macro and inline function b/c it is important to preserve the
        file and line number in the error printing.
    RETURN:
    DEBUG:
    NOTES: 
    FUTURE:
*******************************************************/
#define gpuErrChk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s : %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}



/**********************************
ARGS:
RETURN:
DESCRIPTION:
    Map 2D indices to 1D index
DEBUG:
    1. read_numpy_matrix() uses this function extensively.
       Directly compared output from read_numpy_matrix() with input
       and was IDENTICAL. This could not work if map_idx() didn't 
       function correctly.
FUTURE:
    1. Add error checking if not too expensive
***********************************/
int map_idx(int i, int j, int Ny){
    return (Ny * i + j);
}
// Visible to device
__device__ int d_map_idx(int i, int j, int Ny){
    return (Ny * i + j);
}

/********************************************************
    ARGS:
    DESCRIPTION:
    RETURN:
    DEBUG:
    NOTES: 
    FUTURE:
*******************************************************/
void exit_with_error(char * message){
    fprintf(stderr, "%s", message);
    fflush(stderr);
    exit(1);
}



/**********************************
ARGS:
    path = path to file to read
    dim  = dimension of returned matrix, expected to be len = 2
RETURN:
DESCRIPTION:
    Map 2D indices to 1D index
DEBUG:
    1. Printed out read in matrix. used 'diff' to compare 
       with original. Was IDENTICAL
       --> This function WORKS!
FUTURE:
    1. Add error checking if not too expensive
***********************************/
half * read_numpy_matrix(char* path, int * dim){
    char * line= NULL;
    char * entireFile = NULL;
    char * pch = NULL;  // Used for parsing strings w strtok
    char errStr[500];
    int fileSize = -1;
    int nline = 0;
    int maxchar = 0;    // Maximum number of characters in a lines
    int nchar = 0;      // Number of characters in line
    int ncols = -1;     // Ncolumns in each row.. should be the same for each row
    int ncolsThisRow = 0;   
    int i = 0;
    int j = 0;
    int n = 0;          // Index to loop thru _all_ file chars
    //float * matrix = NULL;
    half * matrix = NULL;
    FILE * f = fopen(path, "r");

    printf("\treading : %s\n", path);
    fflush(stdout);

    //Error check
    if(f == NULL){
        sprintf(errStr, "ERROR!!! %s cannot be opened", path);
        exit_with_error(errStr);
    }
    //Get file size
    fseek(f, 0, SEEK_END);
    fileSize = ftell(f);    // Total num chars in file
    rewind(f);

    //Read entire file
    entireFile = (char* )malloc(sizeof(char) * fileSize);
    fread(entireFile, sizeof(char), fileSize, f);
    rewind(f);

    //Find number of lines and maxchar per line...
    for(n=0; n<fileSize; n++){
        if(entireFile[n] == ' '){
            ncolsThisRow++;
        }
        
        if(entireFile[n] == '\n'){
            maxchar = nchar > maxchar ? nchar : maxchar;

            //Must set at first
            if(nline == 0){
                ncols = ncolsThisRow;
            //Oops, rows aren't the same size.
            }else if(ncols != ncolsThisRow){
                sprintf(errStr, "ERROR!!! nchar %i != ncolsThisRow %i\n", nchar, ncolsThisRow);
                exit_with_error(errStr);
            }
            ncolsThisRow=0;
            nchar = 0;
            nline++;
        }
        nchar++;
    }
    maxchar = maxchar + 1; //+1 for null terminator?
    printf("dim = [nline, ncols] =  [%i, %i],  maxchar = %i \n", nline, ncols, maxchar);
    fflush(stdout);
    
    // Done with busy work - now allocate memory, read in array
    //cudaMallocManaged(&matrix, nline * maxchar * sizeof(float));
    hipMallocManaged(&matrix, nline * maxchar * sizeof(half));
    line   = (char *)malloc(sizeof(char) * maxchar);
    i = 0;
    while(feof(f) == 0){
        if(fgets(line, maxchar, f)){
            //printf("\tEnd of File Reached\n\n");
            //sprintf(errStr, "ERROR!!! in reading 'line'\n");
            //exit_with_error(errStr);
        }
        // Parse line in file
        pch = strtok(line," ");
        j = 0;
        while(pch != NULL){
            //matrix[map_idx(i,j,ncols)] = (float)atof(pch);
            matrix[map_idx(i,j,ncols)] = (half)atof(pch);
            pch = strtok(NULL, " ");
            j++;
        }
        i++;
    }

    /* Debug 
    for(i=0; i<nline; i++){
        for(j=0; j<ncols; j++){
            printf("%.1f ", matrix[map_idx(i,j,ncols)]);
        }
        printf("\n");
    }*/
    
    free(line);
    free(entireFile);
    fclose(f);
    dim[0] = nline;
    dim[1] = ncols;
    return matrix;
}
 


/*************************************************************
ARGS:
    float * A   : 2 x 2 matrix, stored as row majored in 1D array
    int   * dim : len(dim) = 2
RETURN:
    -> newM, is a matrix that is column ordered matrix.
    -> dim is unchanged
DESCRIPTION:
    Take Transpose
DEBUG:
    1. Spot checked beginning, middle and end of matrix. It appears
       that I correctly switched from row-majored to column majored
       matrix
FUTURE:
**************************************************************/
half * reorder_row_major_as_col_major(half * B, int * dim){
    int i,j;    // Indices
    half * newM  = NULL;  //(float *)malloc(sizeof(float) * dim[0] * dim[1]); 
    gpuErrChk(hipMallocManaged(&newM, sizeof(half) * dim[0] * dim[1]));

    //rows
    for(i=0; i<dim[0]; i++){
        for(j=0; j<dim[1]; j++){
            newM[map_idx(j,i,dim[0])]  = B[map_idx(i,j,dim[1])];    // dim[0] or dim[1] for newM?
            //newM[map_idx(i,j,dim[0])]  = B[map_idx(j,i,dim[1])];    // dim[0] or dim[1] for newM?
        }
    }
    printf("Re-ordering matrix B...\n");
    gpuErrChk(hipFree(B));
    return(newM);
}
 

/**********************************
ARGS:
    array1D : 'flattened' 2D array as 1D
    N       : length of array
RETURN:
    N/A
DESCRIPTION:
    Prints 1D array and 3D coords
DEBUG:
    1. spot checked, it works
FUTURE:
***********************************/
void write_1D_array(float * array1D, int Nx, int Ny, FILE * f){
    int i = 0;
    int j = 0;
    int idx = 0;
    for(i=0; i<Nx; i++){
        for(j=0; j<Ny; j++){
            idx = map_idx(i,j,Ny);
            fprintf(f, "%*.1f ", 5, array1D[idx]);
        }
        fprintf(f, "\n");
    }
}


/**********************************
ARGS:
    array1D : 'flattened' 2D array as 1D
    N       : length of array
RETURN:
    N/A
DESCRIPTION:
    Prints 1D array and 3D coords
DEBUG:
    1. spot checked, it works
FUTURE:
***********************************/
void print_1D_array(float * array1D, int Nx, int Ny){
    int i = 0;
    int j = 0;
    int idx = 0;
    for(i=0; i<Nx; i++){
        for(j=0; j<Ny; j++){
            idx = map_idx(i,j,Ny);
            printf("%*.1f ", 5, array1D[idx]);
        }
        printf("\n");
    }
}

/********************************************************
    ARGS:
    DESCRIPTION:
    RETURN:
    DEBUG:
    NOTES: 
        1. Use 'flattened' 2D array
    FUTURE:
*******************************************************/
void initialize_matrix(float *A, int * dim, float value){
    for(int i=0; i<dim[0]; i++){
        for(int j=0; j<dim[1]; j++){
            //A[i*dim[0]+j] = value;
            A[map_idx(i,j,dim[1])] = value;
        }       
    }

}




/********************************************************
    ARGS:
        A : 'flattened' 2d matrix
        B : 'flattened' 2d matrix
        dimA : gives x & y dims
        dimB : gives x & y dims
        dimAB: pointer modified to return size of new matrix

    DESCRIPTION:
        Multiply A*B : Check dims. Expect only 2 dimensions
        for dimA and dimB.
    RETURN:
    DEBUG:
    NOTES: 
        1. blockDim.x  : number of threads in each block
           blockIdx.x  : index of current block
           threadIdx.x : 
        2. Error Check - not possible on device code
    FUTURE:
*******************************************************/
__global__ void matrix_multiply(float * A, float * B, int * dimA, int * dimB,
                                float * AB, int * dimAB)
{
    int j = 0;          // Iterate over elements, do dot product
    int startIdx = blockIdx.x * blockDim.x + threadIdx.x; // Index of current thread in block
    int stride   = blockDim.x * gridDim.x;                // Number of threads in the block
    int ai = 0;         // Index iterating over rows in A
    int bj = 0;         // Index iterating over columns in B
    float sum = 0;
    //printf("%i %i : [%i %i] %i %i\n", startIdx, stride, threadIdx.x, blockIdx.x, blockDim.x, gridDim.x);
    if(blockIdx.x == 0 && threadIdx.x ==0){
        printf("****************************\n\tblockDim.x = %i\n\tgridDim.x = %i\n",
               blockDim.x, gridDim.x);
    }

    // Grid-stride loop
    /**** Row of A to multiply ****/
    for(ai=startIdx; ai<dimA[0]; ai+=stride){       
        //printf("[%i %i] : %i : dimA[0] = %i\n", threadIdx.x, blockIdx.x, ai, dimA[0]);

        /**** Column of AB for output and Columns of B ****/
        for(j=0; j<dimB[1]; j++){ 
            sum = 0;
            for(bj=0; bj<dimB[0]; bj++){
                // EXPENSIVE!! increases runtime 100x
                /*printf("\t[%i, %i] x [%i, %i] = %.0f %.0f\n",
                        ai, bj, j, bj, A[d_map_idx(ai, bj, dimA[1])], B[d_map_idx(j, bj, dimB[0])]); */
                sum += A[d_map_idx(ai, bj, dimA[1])] * B[d_map_idx(j, bj, dimB[0])];
            }
            AB[d_map_idx(ai,j,dimB[1])] = sum;
            //printf("\n");
        }
    }
}



/********************************************************
    ARGS:
        A : 'flattened' 2d matrix
        B : 'flattened' 2d matrix
        dimA : gives x & y dims
        dimB : gives x & y dims
        dimAB: pointer modified to return size of new matrix

    DESCRIPTION:
        Multiply A*B : Check dims. Expect only 2 dimensions
        for dimA and dimB.
    RETURN:
    DEBUG:
    NOTES: 
        1. blockDim.x  : number of threads in each block
           blockIdx.x  : index of current block
           threadIdx.x : 
        2. Error Check - not possible on device code
        3. Taken from : 
            https://devblogs.nvidia.com/programming-tensor-cores-cuda-9/
    FUTURE:
*******************************************************/
__global__ void wmma_example(half * A, half * B, float * C, int M, int N, int K)
{
    //int startIdx = blockIdx.x * blockDim.x + threadIdx.x; // Index of current thread in block
    //int stride   = blockDim.x * gridDim.x;                // Number of threads in the block

    // I DON'T really understand these indices - Figure out later
    int warpSize = blockIdx.x;
    int warpM = (blockIdx.x * blockDim.x + threadIdx.x) / warpSize;     
    int warpN = (blockIdx.y * blockDim.y + threadIdx.y);


    /************
     Fragments for Tensor operations using tensor core.

            C  =      A  *  B 
       (M x N) = (M x K) * (K x N)
    ************/
    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> a_frag;
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> b_frag;
    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> acc_frag;
    //wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> c_frag;
    wmma::fill_fragment(acc_frag, 0.0f);
    

    //printf("%i %i : [%i %i] %i %i\n", startIdx, stride, threadIdx.x, blockIdx.x, blockDim.x, gridDim.x);
    if(blockIdx.x == 0 && threadIdx.x ==0){
        printf("****************************\n\tblockDim.x = %i\n\tgridDim.x = %i\n",
               blockDim.x, gridDim.x);
    }

    for(int k=0; k<K; k+=WMMA_K){
        int aRow = warpM * WMMA_M;
        int aCol = k;   // Recall aCol is contracted with bRow
        int bRow = k;
        int bCol = warpN * WMMA_N;
        int cRow = warpM * WMMA_M;
        int cCol = warpN * WMMA_N;

        // Bounds Checking
        if(aRow < M && aCol < K && bRow < K && bCol<N){

            // Opportunity for disaster here
            //                    (      , memory address, stride (multiple of 4 or 8))
            wmma::load_matrix_sync(a_frag, A + aRow * M + aCol    , M);
            wmma::load_matrix_sync(b_frag, B + bRow     + bCol * K, K);

            // Perform matrix multiplication
            wmma::mma_sync(acc_frag, a_frag, b_frag, acc_frag);

            // Store the output
            wmma::store_matrix_sync(C + cRow * M + cCol, acc_frag, (unsigned)M, wmma::mem_row_major);
        }

    }
}



/********************************************************
    ARGS:
        int argc        : 
        char *argv[]    : 
    DESCRIPTION:
        Can run as 
            ./a.out 
            ./a.out ouputfile
    RETURN:
    DEBUG:
    NOTES: 
    FUTURE:
*******************************************************/
int main(int argc, char *argv[])
{
    // Declare variables
    char path[100];
    char errStr[200];
    int nDev = 0;      //Number of devices
    int * dimA = NULL; //{2,3};
    int * dimB = NULL; //{3,2};
    int * dimAB = NULL; //{0,0};    // Initialize to some value
    half *A = NULL;
    half *B = NULL;
    float *AB = NULL;
    FILE * fout = NULL;
    // Print device statistics.
    hipGetDeviceCount(&nDev);
    for(int i=0; i<nDev; i++){
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printf("Device : %i, Card : %s\n",i,prop.name);
        printf("warpSize : %d\n", prop.warpSize);
        printf("multiProcessorCount : %d\n", prop.multiProcessorCount);
        printf("maxThreadsPerMultiProcessor : %i\n", prop.maxThreadsPerMultiProcessor);
        printf("maxThreadsPerBlock : %i\n", prop.maxThreadsPerBlock);
        printf("maxGridSize : [%i %i %i]\n", prop.maxGridSize[0],
                prop.maxGridSize[1], prop.maxGridSize[2]);
    }


    // This uses CUDA's Unified Memory
    gpuErrChk(hipMallocManaged(&dimA, 2 * sizeof(int)));
    gpuErrChk(hipMallocManaged(&dimB, 2 * sizeof(int)));
    gpuErrChk(hipMallocManaged(&dimAB, 2 * sizeof(int)));
    
    sprintf(path, "data/smaller/A.txt");
    //sprintf(path, "data/very_large/A.txt");
    A = read_numpy_matrix(path, dimA);
    sprintf(path, "data/smaller/B.txt");
    //sprintf(path, "data/very_large/B.txt");
    B = read_numpy_matrix(path, dimB);
    time_t start = time(NULL);
    B = reorder_row_major_as_col_major(B, dimB);
    //sprintf(path, "data/AB_small.txt");
    //sprintf(path, "data/large/AB.txt");
    //answer = read_numpy_matrix(path, dimAB);

    // Try CUDA version of matrix_multiply
    dimAB[0] = dimA[0];
    dimAB[1] = dimB[1];
    gpuErrChk(hipMallocManaged(&AB, dimAB[0] * dimAB[1] * sizeof(float)));
    //            <<<gridDim.x (# blocks), blockDim.x (# threads per block) >>>

    wmma_example<<<1,32>>> (A, B, AB, dimA[0], dimB[1], dimA[1]);  // Fails b/c maxThreadsPerBlock=1024
    //matrix_multiply<<<2,3>>> (A, B, dimA, dimB, AB, dimAB);  // Fails b/c maxThreadsPerBlock=1024
    gpuErrChk(hipPeekAtLastError());
    gpuErrChk(hipDeviceSynchronize());

    printf("Run time : %.3f s\n", difftime(time(NULL), start));
    if(argc == 1){
        fout = fopen("output/AB_result.txt", "w+");
    }else if(argc == 2){
        fout = fopen(argv[1], "w+");
    }else{
        sprintf(errStr, "ERROR!!! Incorrect number of arguments");
        exit_with_error(errStr);
    }
    write_1D_array(AB, dimAB[0], dimAB[1], fout);
    fclose(fout);


    return 0;
}

