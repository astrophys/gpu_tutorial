//Compile : nvcc -g -G -arch=sm_70 src/understanding_cuda_gdb.cu

#include <hip/hip_runtime.h>
#include <string.h>
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <mma.h>
using namespace nvcuda; 
using namespace std; 

#define gpuErrChk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s : %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


__global__ void print_hello()
{
    printf("Hello from : blockIdx = [%i %i], threadIdx = [%i %i]\n",
           blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y);
}

int main(int argc, char *argv[])
{
    dim3 blockD(4,4);     // threads
    dim3 gridD(2,2);      // blocks
    
    print_hello <<<gridD,blockD>>> ();
    gpuErrChk(hipPeekAtLastError());
    gpuErrChk(hipDeviceSynchronize());

    print_hello <<<gridD,blockD>>> ();
    gpuErrChk(hipPeekAtLastError());
    gpuErrChk(hipDeviceSynchronize());

    return 0;
}
