/*
Author : Ali Snedden
Date   : 11/28/18
Purpose:
    This is a program that multiplies two matrices. This combines features from 
    matrix_multiply_omp_cache_optimized.c (e.g. taking the transpose of B) and using
    CUDA.
Debug  : 
Notes  : 
    1. How to Run:
        module load cuda/8.0 
        nvcc matrix_multiply.cu    
    2. http://developer.download.nvidia.com/compute/cuda/3_1/toolkit/docs/NVIDIA_CUDA_C_ProgrammingGuide_3.1.pdf
    3. Unified Memory : https://devblogs.nvidia.com/unified-memory-cuda-beginners/
    4. Cannot kill errant threads and cleanly end computation in CUDA
       --> See : https://stackoverflow.com/questions/52116815/is-there-a-way-terminate-host-and-device-program-execution-if-a-cuda-thread-enco
    5. 'Proper' error handling : https://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api
    6. Recall that we can't print to stderr from gpu thread
    7. To debug cuda-gdb ./a.out
    8. Biggest error is in forgetting to allocate memory between the device and host,
       e.g. cudaMallocManaged()
    9. IO is very expensive.  Appears to get flushed on cudaDeviceSynchronize()
    10. Using matrix_multiply<<<1,1>>> == 431s, while cpu version 14s. Clearly there
        is substantial overhead when using running single gpu thread.
    11. If using more than maxThreadsPerBlock, it fails to compute and doesn't emit an 
        error.
        --> After each kernel call, do gpuErrchk( cudaPeekAtLastError() );

Good Weblinks:
    1. Unified Memory : https://devblogs.nvidia.com/unified-memory-cuda-beginners/

Future :
    1. Try managing memory directly on Host and Device.

*/

#include <hip/hip_runtime.h>
#include <iostream>
#include <string.h>
#include <stdio.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime_api.h>

// This is C++ code - from stackoverflow : https://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api
/********************************************************
    ARGS:
        cudaError_t code
        const char* file : 
        int line :
    DESCRIPTION:
        Uses macro and inline function b/c it is important to preserve the
        file and line number in the error printing.
    RETURN:
    DEBUG:
    NOTES: 
    FUTURE:
*******************************************************/
#define gpuErrChk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s : %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}



/**********************************
ARGS:
RETURN:
DESCRIPTION:
    Map 2D indices to 1D index
DEBUG:
    1. read_numpy_matrix() uses this function extensively.
       Directly compared output from read_numpy_matrix() with input
       and was IDENTICAL. This could not work if map_idx() didn't 
       function correctly.
FUTURE:
    1. Add error checking if not too expensive
***********************************/
int map_idx(int i, int j, int Ny){
    return (Ny * i + j);
}
__device__ int d_map_idx(int i, int j, int Ny){
    return (Ny * i + j);
}

/********************************************************
    ARGS:
    DESCRIPTION:
    RETURN:
    DEBUG:
    NOTES: 
    FUTURE:
*******************************************************/
void exit_with_error(char * message){
    fprintf(stderr, "%s", message);
    fflush(stderr);
    exit(1);
}
//__device__ void d_exit_with_error(char * message){
//    fprintf(stderr, "%s", message);
//    fflush(stderr);
//    exit(1);
//}



/**********************************
ARGS:
    path = path to file to read
    dim  = dimension of returned matrix, expected to be len = 2
RETURN:
DESCRIPTION:
    Map 2D indices to 1D index
DEBUG:
    1. Printed out read in matrix. used 'diff' to compare 
       with original. Was IDENTICAL
       --> This function WORKS!
FUTURE:
    1. Add error checking if not too expensive
***********************************/
float * read_numpy_matrix(char* path, int * dim){
    char * line= NULL;
    char * entireFile = NULL;
    char * pch = NULL;  // Used for parsing strings w strtok
    char errStr[500];
    int fileSize = -1;
    int nline = 0;
    int maxchar = 0;    // Maximum number of characters in a lines
    int nchar = 0;      // Number of characters in line
    int ncols = -1;     // Ncolumns in each row.. should be the same for each row
    int ncolsThisRow = 0;   
    int i = 0;
    int j = 0;
    int n = 0;          // Index to loop thru _all_ file chars
    float * matrix = NULL;
    FILE * f = fopen(path, "r");

    printf("\treading : %s\n", path);
    fflush(stdout);

    //Error check
    if(f == NULL){
        sprintf(errStr, "ERROR!!! %s cannot be opened", path);
        exit_with_error(errStr);
    }
    //Get file size
    fseek(f, 0, SEEK_END);
    fileSize = ftell(f);    // Total num chars in file
    rewind(f);

    //Read entire file
    entireFile = (char* )malloc(sizeof(char) * fileSize);
    fread(entireFile, sizeof(char), fileSize, f);
    rewind(f);

    //Find number of lines and maxchar per line...
    for(n=0; n<fileSize; n++){
        if(entireFile[n] == ' '){
            ncolsThisRow++;
        }
        
        if(entireFile[n] == '\n'){
            maxchar = nchar > maxchar ? nchar : maxchar;

            //Must set at first
            if(nline == 0){
                ncols = ncolsThisRow;
            //Oops, rows aren't the same size.
            }else if(ncols != ncolsThisRow){
                sprintf(errStr, "ERROR!!! nchar %i != ncolsThisRow %i\n", nchar, ncolsThisRow);
                exit_with_error(errStr);
            }
            ncolsThisRow=0;
            nchar = 0;
            nline++;
        }
        nchar++;
    }
    maxchar = maxchar + 1; //+1 for null terminator?
    printf("dim = [nline, ncols] =  [%i, %i],  maxchar = %i \n", nline, ncols, maxchar);
    fflush(stdout);
    
    // Done with busy work - now allocate memory, read in array
    hipMallocManaged(&matrix, nline * maxchar * sizeof(float));
    line   = (char *)malloc(sizeof(char) * maxchar);
    i = 0;
    while(feof(f) == 0){
        if(fgets(line, maxchar, f)){
            //printf("\tEnd of File Reached\n\n");
            //sprintf(errStr, "ERROR!!! in reading 'line'\n");
            //exit_with_error(errStr);
        }
        // Parse line in file
        pch = strtok(line," ");
        j = 0;
        while(pch != NULL){
            matrix[map_idx(i,j,ncols)] = (float)atof(pch);
            pch = strtok(NULL, " ");
            j++;
        }
        i++;
    }

    /* Debug 
    for(i=0; i<nline; i++){
        for(j=0; j<ncols; j++){
            printf("%.1f ", matrix[map_idx(i,j,ncols)]);
        }
        printf("\n");
    }*/
    
    free(line);
    free(entireFile);
    fclose(f);
    dim[0] = nline;
    dim[1] = ncols;
    return matrix;
}
 


/*************************************************************
ARGS:
    float * A   : 2 x 2 matrix, stored as row majored in 1D array
    int   * dim : len(dim) = 2
RETURN:
    -> newM, is a matrix that is column ordered matrix.
    -> dim is unchanged
DESCRIPTION:
    Take Transpose
DEBUG:
    1. Spot checked beginning, middle and end of matrix. It appears
       that I correctly switched from row-majored to column majored
       matrix
FUTURE:
**************************************************************/
float * reorder_row_major_as_col_major(float * B, int * dim){
    int i,j;    // Indices
    float * newM  = NULL;  //(float *)malloc(sizeof(float) * dim[0] * dim[1]); 
    gpuErrChk(hipMallocManaged(&newM, sizeof(float) * dim[0] * dim[1]));

    //rows
    for(i=0; i<dim[0]; i++){
        for(j=0; j<dim[1]; j++){
            newM[map_idx(j,i,dim[0])]  = B[map_idx(i,j,dim[1])];    // dim[0] or dim[1] for newM?
            //newM[map_idx(i,j,dim[0])]  = B[map_idx(j,i,dim[1])];    // dim[0] or dim[1] for newM?
        }
    }
    printf("Re-ordering matrix B...\n");
    gpuErrChk(hipFree(B));
    return(newM);
}
 

/**********************************
ARGS:
    array1D : 'flattened' 2D array as 1D
    N       : length of array
RETURN:
    N/A
DESCRIPTION:
    Prints 1D array and 3D coords
DEBUG:
    1. spot checked, it works
FUTURE:
***********************************/
void write_1D_array(float * array1D, int Nx, int Ny, FILE * f){
    int i = 0;
    int j = 0;
    int idx = 0;
    for(i=0; i<Nx; i++){
        for(j=0; j<Ny; j++){
            idx = map_idx(i,j,Ny);
            fprintf(f, "%*.1f ", 5, array1D[idx]);
        }
        fprintf(f, "\n");
    }
}


/**********************************
ARGS:
    array1D : 'flattened' 2D array as 1D
    N       : length of array
RETURN:
    N/A
DESCRIPTION:
    Prints 1D array and 3D coords
DEBUG:
    1. spot checked, it works
FUTURE:
***********************************/
void print_1D_array(float * array1D, int Nx, int Ny){
    int i = 0;
    int j = 0;
    int idx = 0;
    for(i=0; i<Nx; i++){
        for(j=0; j<Ny; j++){
            idx = map_idx(i,j,Ny);
            printf("%*.1f ", 5, array1D[idx]);
        }
        printf("\n");
    }
}

/********************************************************
    ARGS:
    DESCRIPTION:
    RETURN:
    DEBUG:
    NOTES: 
        1. Use 'flattened' 2D array
    FUTURE:
*******************************************************/
void initialize_matrix(float *A, int * dim, float value){
    for(int i=0; i<dim[0]; i++){
        for(int j=0; j<dim[1]; j++){
            //A[i*dim[0]+j] = value;
            A[map_idx(i,j,dim[1])] = value;
        }       
    }

}




/********************************************************
    ARGS:
        A : 'flattened' 2d matrix
        B : 'flattened' 2d matrix
        dimA : gives x & y dims
        dimB : gives x & y dims
        dimAB: pointer modified to return size of new matrix

    DESCRIPTION:
        Multiply A*B : Check dims. Expect only 2 dimensions
        for dimA and dimB.
    RETURN:
    DEBUG:
    NOTES: 
        1. blockDim.x  : number of threads in each block
           blockIdx.x  : index of current block
           threadIdx.x : 
        2. Error Check - not possible on device code
    FUTURE:
*******************************************************/
__global__ void matrix_multiply(float * A, float * B, int * dimA, int * dimB,
                                float * AB, int * dimAB)
{
    int j = 0;          // Iterate over elements, do dot product
    int startIdx = blockIdx.x * blockDim.x + threadIdx.x; // Index of current thread in block
    int stride   = blockDim.x * gridDim.x;                // Number of threads in the block
    int ai = 0;         // Index iterating over rows in A
    int bj = 0;         // Index iterating over columns in B
    float sum = 0;
    //printf("%i %i : [%i %i] %i %i\n", startIdx, stride, threadIdx.x, blockIdx.x, blockDim.x, gridDim.x);
    if(blockIdx.x == 0 && threadIdx.x ==0){
        printf("****************************\n\tblockDim.x = %i\n\tgridDim.x = %i\n",
               blockDim.x, gridDim.x);
    }

    // if(dimA[1] != dimB[0]){
    //     char errStr[] = "ERROR!! dimension mismatch\n";
    //     //sprintf(errStr, "ERROR!! dimension mismatch, %i != %i", dimA[1], dimB[0]);
    //     d_exit_with_error(errStr);
    // }
    
    // Grid-stride loop
    /**** Row of A to multiply ****/
    for(ai=startIdx; ai<dimA[0]; ai+=stride){       
        //printf("[%i %i] : %i : dimA[0] = %i\n", threadIdx.x, blockIdx.x, ai, dimA[0]);

        /**** Column of AB for output and Columns of B ****/
        for(j=0; j<dimB[1]; j++){ 
            sum = 0;
            for(bj=0; bj<dimB[0]; bj++){
                // EXPENSIVE!! increases runtime 100x
                /*printf("\t[%i, %i] x [%i, %i] = %.0f %.0f\n",
                        ai, bj, j, bj, A[d_map_idx(ai, bj, dimA[1])], B[d_map_idx(j, bj, dimB[0])]); */
                sum += A[d_map_idx(ai, bj, dimA[1])] * B[d_map_idx(j, bj, dimB[0])];
            }
            AB[d_map_idx(ai,j,dimB[1])] = sum;
            //printf("\n");
        }
    }
}





/********************************************************
    ARGS:
        int argc        : 
        char *argv[]    : 
    DESCRIPTION:
        Can run as 
            ./a.out 
            ./a.out ouputfile
    RETURN:
    DEBUG:
    NOTES: 
    FUTURE:
*******************************************************/
int main(int argc, char *argv[])
{
    // Declare variables
    char path[100];
    char errStr[200];
    int nDev = 0;      //Number of devices
    int * dimA = NULL; //{2,3};
    int * dimB = NULL; //{3,2};
    int * dimAB = NULL; //{0,0};    // Initialize to some value
    float *A = NULL;
    float *B = NULL;
    float *AB = NULL;
    float *answer = NULL;
    FILE * fout = NULL;
    // Print device statistics.
    hipGetDeviceCount(&nDev);
    for(int i=0; i<nDev; i++){
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printf("Device : %i, Card : %s\n",i,prop.name);
        printf("warpSize : %d\n", prop.warpSize);
        printf("multiProcessorCount : %d\n", prop.multiProcessorCount);
        printf("maxThreadsPerMultiProcessor : %i\n", prop.maxThreadsPerMultiProcessor);
        printf("maxThreadsPerBlock : %i\n", prop.maxThreadsPerBlock);
        printf("maxGridSize : %i\n", prop.maxGridSize);
    }


    // This uses CUDA's Unified Memory
    gpuErrChk(hipMallocManaged(&dimA, 2 * sizeof(float)));
    gpuErrChk(hipMallocManaged(&dimB, 2 * sizeof(float)));
    gpuErrChk(hipMallocManaged(&dimAB, 2 * sizeof(float)));
    
    //sprintf(path, "data/very_small/A.txt");
    sprintf(path, "data/large/A.txt");
    A = read_numpy_matrix(path, dimA);
    //sprintf(path, "data/very_small/B.txt");
    sprintf(path, "data/large/B.txt");
    B = read_numpy_matrix(path, dimB);
    time_t start = time(NULL);
    B = reorder_row_major_as_col_major(B, dimB);
    //sprintf(path, "data/AB_small.txt");
    //sprintf(path, "data/large/AB.txt");
    //answer = read_numpy_matrix(path, dimAB);

    // Try CUDA version of matrix_multiply
    dimAB[0] = dimA[0];
    dimAB[1] = dimB[1];
    gpuErrChk(hipMallocManaged(&AB, dimAB[0] * dimAB[1] * sizeof(float)));
    //            <<<gridDim.x (# blocks), blockDim.x (# threads per block) >>>

    matrix_multiply<<<1024,32>>> (A, B, dimA, dimB, AB, dimAB);  // Fails b/c maxThreadsPerBlock=1024
    //matrix_multiply<<<2,3>>> (A, B, dimA, dimB, AB, dimAB);  // Fails b/c maxThreadsPerBlock=1024
    gpuErrChk( hipPeekAtLastError());
    gpuErrChk(hipDeviceSynchronize());

    printf("Run time : %.3f s\n", difftime(time(NULL), start));
    if(argc == 1){
        fout = fopen("output/AB_result.txt", "w+");
    }else if(argc == 2){
        fout = fopen(argv[1], "w+");
    }else{
        sprintf(errStr, "ERROR!!! Incorrect number of arguments");
        exit_with_error(errStr);
    }
    write_1D_array(AB, dimAB[0], dimAB[1], fout);
    fclose(fout);


    return 0;
}

